#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include ""
#define blockSize 128


namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
		__global__ void upSweep(int n, int k, int *dev) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) return;

			if ((index % (2 * k) == 0) && (index + (2 * k) <= n))
				dev[index + (2 * k) - 1] += dev[index + k - 1];
		}

		__global__ void downSweep(int n, int k, int *idata) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) return;
			// need to check boundary
			if ((index % (2 * k) == 0) && (index + (2 * k) <= n)) {
				int temp = idata[index + k - 1];
				idata[index + k - 1] = idata[index + (2 * k) - 1];
				idata[index + (2 * k) - 1] += temp;
			}
		}

        void scan(int n, int *odata, const int *idata) {
			int *exclusive;
			int length = pow(2, ilog2ceil(n));
			hipMalloc((int**)&exclusive, length * sizeof(int));
			hipMemset(exclusive, 0, length * sizeof(int));
			hipMemcpy(exclusive, idata, n * sizeof(int), hipMemcpyHostToDevice);
			dim3 fullBlocksPerGrid((length + blockSize - 1) / blockSize);
			timer().startGpuTimer();
			// TODO
			// up-sweep
			for (int d = 0; d < ilog2ceil(length); d++) {
				upSweep<<< fullBlocksPerGrid, blockSize >>>(length, pow(2, d), exclusive);
			}
			hipMemset(exclusive + length - 1, 0, sizeof(int));
			// down-sweep
			for (int d = ilog2ceil(length) - 1; d >= 0; d--) {
				downSweep<<< fullBlocksPerGrid, blockSize >>>(length, pow(2, d), exclusive);
			}
            timer().endGpuTimer();
			hipMemcpy(odata, exclusive, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(exclusive);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
